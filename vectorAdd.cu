#include <iostream>
#include <hip/hip_runtime.h>
 
#define N 16384
 
// write kernel function of vector addition
__global__ void vecAdd(float *a, float *b, float *c, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}
 
int main()
{
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
    int size = N * sizeof(float);
 
    // allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
 
    // allocate space for host copies of a, b, c and setup input values
    a = (float *)malloc(size);
    b = (float *)malloc(size);
    c = (float *)malloc(size);
 
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i * i;
    }
 
    // copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
 
    // launch vecAdd() kernel on GPU
    vecAdd<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);
 
    hipDeviceSynchronize();
 
    // copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
 
    // verify result
    for (int i = 0; i < N; i++)
    {
        if (a[i] + b[i] != c[i])
        {
            std::cout << "Error: " << a[i] << " + " << b[i] << " != " << c[i] << std::endl;
            break;
        }
    }
 
    std::cout << "Done!" << std::endl;
 
    // clean up
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    return 0;
}